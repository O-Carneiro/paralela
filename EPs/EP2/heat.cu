#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>

#define WALL_TEMP 20.0
#define FIREPLACE_TEMP 100.0

#define FIREPLACE_START 3
#define FIREPLACE_END 7
#define ROOM_SIZE 10

int threads_per_block;
int blocks_in_grid;

typedef struct {
	int blockX;
	int blockY;
	int gridX;
	int gridY;
} cudaData;

void initialize(double **h, int n)
{
    int fireplace_start = (FIREPLACE_START * n) / ROOM_SIZE;
    int fireplace_end = (FIREPLACE_END * n) / ROOM_SIZE;

    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            if (i == 0 || i == n - 1 || j == 0 || j == n - 1)
            {
                h[i][j] = (i == n - 1 && j >= fireplace_start && j <= fireplace_end) ? FIREPLACE_TEMP : WALL_TEMP;
            }
            else
            {
                h[i][j] = 0.0;
            }
        }
    }
}

void jacobi_iteration_host(double **h, double **g, int n, int iter_limit)
{
    for (int iter = 0; iter < iter_limit; iter++)
    {
        for (int i = 1; i < n - 1; i++)
        {
            for (int j = 1; j < n - 1; j++)
            {
                g[i][j] = 0.25 * (h[i - 1][j] + h[i + 1][j] + h[i][j - 1] + h[i][j + 1]);
            }
        }
        for (int i = 1; i < n - 1; i++)
        {
            for (int j = 1; j < n - 1; j++)
            {
                h[i][j] = g[i][j];
            }
        }
    }
}

double calculate_elapsed_time(struct timespec start, struct timespec end)
{
    double start_sec = (double)start.tv_sec * 1e9 + (double)start.tv_nsec;
    double end_sec = (double)end.tv_sec * 1e9 + (double)end.tv_nsec;
    return (end_sec - start_sec) / 1e9;
}

void save_to_file(double **h, int n, char *filename)
{
    FILE *file = fopen(filename, "w");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            fprintf(file, "%lf ", h[i][j]);
        }
        fprintf(file, "\n");
    }
    fclose(file);
}

__global__ void jacobi_kernel(double *d_h, double *d_g, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > 0 && i < n - 1 && j > 0 && j < n - 1) {
        d_g[i * n + j] = 0.25 * (d_h[(i - 1) * n + j] + d_h[(i + 1) * n + j] +
                                 d_h[i * n + (j - 1)] + d_h[i * n + (j + 1)]);
    }

}

void jacobi_iteration_cu(double **h, double **g, int n, int iter_limit, cudaData arg) {
    double *h_flat = (double *)malloc(n * n * sizeof(double));
    double *g_flat = (double *)malloc(n * n * sizeof(double));
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            h_flat[i * n + j] = h[i][j];
            g_flat[i * n + j] = g[i][j];
        }
    }

    double *d_h, *d_g;
    hipMalloc(&d_h, n * n * sizeof(double));
    hipMalloc(&d_g, n * n * sizeof(double));
    hipMemcpy(d_h, h_flat, n * n * sizeof(double), hipMemcpyHostToDevice);

	dim3 blockDim(arg.blockX, arg.blockY);
	dim3 gridDim(arg.gridX, arg.gridY);

    for (int iter = 0; iter < iter_limit; iter++) {
        jacobi_kernel<<<gridDim, blockDim>>>(d_h, d_g, n);
        hipDeviceSynchronize();
        double *temp = d_h;
        d_h = d_g;
        d_g = temp;
    }
    hipMemcpy(h_flat, d_h, n * n * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            h[i][j] = h_flat[i * n + j];
        }
    }
    free(h_flat);
    free(g_flat);
    hipFree(d_h);
    hipFree(d_g);
}

int main(int argc, char *argv[])
{
    if (argc < 7) {
        fprintf(stderr, "Usage: %s <number of points> <iteration limit> <block_x> <block_y> <grid_x> <grid_y>\n", argv[0]);
        return 1;
    }

    int n = atoi(argv[1]);
    int iter_limit = atoi(argv[2]);
    int block_x = atoi(argv[3]);
    int block_y = atoi(argv[4]);
    int grid_x = atoi(argv[5]);
    int grid_y = atoi(argv[6]);

    // Validate block size
    if (block_x <= 0 || block_y <= 0) {
        fprintf(stderr, "Block dimensions must be positive integers.\n");
        return 1;
    }

    // Validate grid size
    if (grid_x <= 0 || grid_y <= 0) {
        fprintf(stderr, "Grid dimensions must be positive integers.\n");
        return 1;
    }

    int total_threads = grid_x * grid_y * block_x * block_y;
    if (total_threads < n*n) {
        fprintf(stderr, "Warning: Not enough threads to cover the problem size.\n");
        return 1;
    }
	cudaData arg;
	arg.blockX = block_x;
	arg.blockY = block_y;
	arg.gridX = grid_x;
	arg.gridY = grid_y;

    double **h = (double **)malloc(n * sizeof(double *));
    double **g = (double **)malloc(n * sizeof(double *));
    if (h == NULL || g == NULL)
    {
        fprintf(stderr, "Erro ao alocar memória para h ou g\n");
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < n; i++)
    {
        h[i] = (double *)malloc(n * sizeof(double));
        g[i] = (double *)malloc(n * sizeof(double));
        if (h[i] == NULL || g[i] == NULL)
        {
            fprintf(stderr, "Erro ao alocar memória para h[%d] ou g[%d]\n", i, i);
            exit(EXIT_FAILURE);
        }
    }

    struct timespec start, end;
    initialize(h, n);
    char filename[256] = "device.txt";

    clock_gettime(CLOCK_MONOTONIC, &start);
    jacobi_iteration_cu(h, g, n, iter_limit, arg);
    clock_gettime(CLOCK_MONOTONIC, &end);
    save_to_file(h, n, filename);

    double elapsed_time_d = calculate_elapsed_time(start, end);
    printf("%.9f,", elapsed_time_d);

    initialize(h, n);

    strcpy(filename, "host.txt");

    clock_gettime(CLOCK_MONOTONIC, &start);
    jacobi_iteration_host(h, g, n, iter_limit);
    clock_gettime(CLOCK_MONOTONIC, &end);
    save_to_file(h, n, filename);

    double elapsed_time_h = calculate_elapsed_time(start, end);
    printf("%.9f,", elapsed_time_h);
	printf("%.9f\n", elapsed_time_h/elapsed_time_d);

    for (int i = 0; i < n; i++)
    {
        free(h[i]);
        free(g[i]);
    }
    free(h);
    free(g);

    return 0;
}
