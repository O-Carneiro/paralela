#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>

#define WALL_TEMP 20.0
#define FIREPLACE_TEMP 100.0
#define BODY_TEMP 37.0 // Temperatura do corpo
#define FIREPLACE_START 3
#define FIREPLACE_END 7
#define ROOM_SIZE 10

typedef struct {
    int blockX;
    int blockY;
    int gridX;
    int gridY;
} cudaData;

void initialize(double **matrix, int n, int body_start, int body_end) {
    int fireplace_start = (FIREPLACE_START * n) / ROOM_SIZE;
    int fireplace_end = (FIREPLACE_END * n) / ROOM_SIZE;

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == 0 || i == n - 1 || j == 0 || j == n - 1) {
                matrix[i][j] = (i == n - 1 && j >= fireplace_start && j <= fireplace_end) ? FIREPLACE_TEMP : WALL_TEMP;
            } else if (i >= body_start && i <= body_end && j >= body_start && j <= body_end) {
                matrix[i][j] = BODY_TEMP;
            } else {
                matrix[i][j] = 0.0;
            }
        }
    }
}

void jacobi_iteration_host(double **h, double **g, int n, int iter_limit, int body_start, int body_end) {
    for (int iter = 0; iter < iter_limit; iter++) {
        for (int i = 1; i < n - 1; i++) {
            for (int j = 1; j < n - 1; j++) {
                if (!(i >= body_start && i <= body_end && j >= body_start && j <= body_end)) {
                    g[i][j] = 0.25 * (h[i - 1][j] + h[i + 1][j] + h[i][j - 1] + h[i][j + 1]);
                }
            }
        }
        for (int i = 1; i < n - 1; i++) {
            for (int j = 1; j < n - 1; j++) {
                if (!(i >= body_start && i <= body_end && j >= body_start && j <= body_end)) {
                    h[i][j] = g[i][j];
                }
            }
        }
    }
}

__global__ void jacobi_kernel(double *d_h, double *d_g, int n, int body_start, int body_end) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (i > 0 && i < n - 1 && j > 0 && j < n - 1) {
        if (!(i >= body_start && i <= body_end && j >= body_start && j <= body_end)) {
            d_g[i * n + j] = 0.25 * (d_h[(i - 1) * n + j] + d_h[(i + 1) * n + j] +
                                     d_h[i * n + (j - 1)] + d_h[i * n + (j + 1)]);
        }
    }
}

__global__ void g_to_h(double *d_h, double *d_g, int n, int body_start, int body_end) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (i > 0 && i < n - 1 && j > 0 && j < n - 1) {
        if (!(i >= body_start && i <= body_end && j >= body_start && j <= body_end)) {
            d_h[i * n + j] = d_g[i * n + j];
        }
    }
}

void jacobi_iteration_cu(double **h, double **g, int n, int iter_limit, cudaData arg, int body_start, int body_end) {
    double *h_flat = (double *)malloc(n * n * sizeof(double));
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            h_flat[i * n + j] = h[i][j];
        }
    }

    double *d_h, *d_g;
    hipMalloc(&d_h, n * n * sizeof(double));
    hipMalloc(&d_g, n * n * sizeof(double));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Transfer data to device
    hipEventRecord(start, 0);
    hipMemcpy(d_h, h_flat, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Data transfer to device: %.2f ms\n", elapsedTime);

    dim3 blockDim(arg.blockX, arg.blockY);
    dim3 gridDim(arg.gridX, arg.gridY);

    // Run Jacobi kernel
    hipEventRecord(start, 0);
    for (int iter = 0; iter < iter_limit; iter++) {
        jacobi_kernel<<<gridDim, blockDim>>>(d_h, d_g, n, body_start, body_end);
        hipDeviceSynchronize();
        g_to_h<<<gridDim, blockDim>>>(d_h, d_g, n, body_start, body_end);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Kernel execution: %.2f ms\n", elapsedTime);

    // Transfer data back to host
    hipEventRecord(start, 0);
    hipMemcpy(h_flat, d_h, n * n * sizeof(double), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Data transfer to host: %.2f ms\n", elapsedTime);

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            h[i][j] = h_flat[i * n + j];
        }
    }

    free(h_flat);
    hipFree(d_h);
    hipFree(d_g);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

bool compare_results(double **host_result, double **device_result, int n, double tolerance) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (fabs(host_result[i][j] - device_result[i][j]) > tolerance) {
                printf("Mismatch at (%d, %d): host = %lf, device = %lf\n", i, j, host_result[i][j], device_result[i][j]);
                return false;
            }
        }
    }
    return true;
}

double calculate_elapsed_time(struct timespec start, struct timespec end) {
    double start_sec = (double)start.tv_sec * 1e9 + (double)start.tv_nsec;
    double end_sec = (double)end.tv_sec * 1e9 + (double)end.tv_nsec;
    return (end_sec - start_sec) / 1e9;
}

void save_to_file(double **matrix, int n, const char *filename) {
    FILE *file = fopen(filename, "w");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            fprintf(file, "%lf ", matrix[i][j]);
        }
        fprintf(file, "\n");
    }
    fclose(file);
}

int main(int argc, char *argv[]) {
    if (argc < 7) {
        fprintf(stderr, "Usage: %s <number of points> <iteration limit> <block_x> <block_y> <grid_x> <grid_y>\n", argv[0]);
        return 1;
    }

    int n = atoi(argv[1]);
    int iter_limit = atoi(argv[2]);
    int block_x = atoi(argv[3]);
    int block_y = atoi(argv[4]);
    int grid_x = atoi(argv[5]);
    int grid_y = atoi(argv[6]);

    // Validate block and grid sizes
    if (block_x <= 0 || block_y <= 0 || grid_x <= 0 || grid_y <= 0) {
        fprintf(stderr, "Block and grid dimensions must be positive integers.\n");
        return 1;
    }

    // Define the body hot area
    int body_start = n / 4;
    int body_end = n / 4 + n / 10;

    double **h_host = (double **)malloc(n * sizeof(double *));
    double **g_host = (double **)malloc(n * sizeof(double *));
    double **h_device = (double **)malloc(n * sizeof(double *));
    for (int i = 0; i < n; i++) {
        h_host[i] = (double *)malloc(n * sizeof(double));
        g_host[i] = (double *)malloc(n * sizeof(double));
        h_device[i] = (double *)malloc(n * sizeof(double));
        if (!h_host[i] || !g_host[i] || !h_device[i]) {
            fprintf(stderr, "Error allocating memory for matrices\n");
            exit(EXIT_FAILURE);
        }
    }

    cudaData arg = { block_x, block_y, grid_x, grid_y };

    struct timespec start, end;
    initialize(h_host, n, body_start, body_end);
    initialize(h_device, n, body_start, body_end);

    // GPU computation
    clock_gettime(CLOCK_MONOTONIC, &start);
    jacobi_iteration_cu(h_device, g_host, n, iter_limit, arg, body_start, body_end);
    clock_gettime(CLOCK_MONOTONIC, &end);
    double elapsed_time_d = calculate_elapsed_time(start, end);
    printf("GPU Total time: %.9f seconds\n", elapsed_time_d);
    save_to_file(h_device, n, "device.txt");

    // CPU computation
    clock_gettime(CLOCK_MONOTONIC, &start);
    jacobi_iteration_host(h_host, g_host, n, iter_limit, body_start, body_end);
    clock_gettime(CLOCK_MONOTONIC, &end);
    double elapsed_time_h = calculate_elapsed_time(start, end);
    printf("CPU Total time: %.9f seconds\n", elapsed_time_h);
    save_to_file(h_host, n, "host.txt");

    // Compare results
    if (compare_results(h_host, h_device, n, 1e-6)) {
        printf("The results match within the tolerance.\n");
    } else {
        printf("The results do not match!\n");
    }

    printf("Speedup (CPU/GPU): %.2f\n", elapsed_time_h / elapsed_time_d);

    for (int i = 0; i < n; i++) {
        free(h_host[i]);
        free(g_host[i]);
        free(h_device[i]);
    }
    free(h_host);
    free(g_host);
    free(h_device);

    return 0;
}
